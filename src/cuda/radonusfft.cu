#include "hip/hip_runtime.h"
#include <stdio.h>

#include "radonusfft.cuh"
#include "kernels.cu"
#include "shift.cu"

radonusfft::radonusfft(size_t ntheta, size_t pnz, size_t n, float center,
                       size_t theta_)
    : ntheta(ntheta), pnz(pnz), n(n), center(center) {
  float eps = 1e-3;
  mu = -log(eps) / (2 * n * n);
  m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4));
  hipMalloc((void **)&f, n * n * pnz * sizeof(float2));
  hipMalloc((void **)&g, n * ntheta * pnz * sizeof(float2));
  hipMalloc((void **)&fde, 2 * n * 2 * n * pnz * sizeof(float2));
  hipMalloc((void **)&fdee,
             (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

  hipMalloc((void **)&x, n * ntheta * sizeof(float));
  hipMalloc((void **)&y, n * ntheta * sizeof(float));
  hipMalloc((void **)&theta, ntheta * sizeof(float));
  hipMemcpy(theta, (float *)theta_, ntheta * sizeof(float), hipMemcpyDefault);

  int ffts[2];
  int idist;
  int odist;
  int inembed[2];
  int onembed[2];
  // fft 2d
  ffts[0] = 2 * n;
  ffts[1] = 2 * n;
  idist = 2 * n * 2 * n;
  odist = (2 * n + 2 * m) * (2 * n + 2 * m);
  inembed[0] = 2 * n;
  inembed[1] = 2 * n;
  onembed[0] = 2 * n + 2 * m;
  onembed[1] = 2 * n + 2 * m;
  hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist,
                HIPFFT_C2C, pnz);
  hipfftPlanMany(&plan2dadj, 2, ffts, onembed, 1, odist, inembed, 1, idist,
                HIPFFT_C2C, pnz);

  // fft 1d
  ffts[0] = n;
  idist = n;
  odist = n;
  inembed[0] = n;
  onembed[0] = n;
  hipfftPlanMany(&plan1d, 1, ffts, inembed, 1, idist, onembed, 1, odist,
                HIPFFT_C2C, ntheta * pnz);
  hipMalloc((void **)&shiftfwd, n * sizeof(float2));
  hipMalloc((void **)&shiftadj, n * sizeof(float2));
  // compute shifts with respect to the rotation center
  takeshift <<<ceil(n / 1024.0), 1024>>> (shiftfwd, -(center - n / 2.0), n);
  takeshift <<<ceil(n / 1024.0), 1024>>> (shiftadj, (center - n / 2.0), n);

  BS2d = dim3(32, 32);
  BS3d = dim3(32, 32, 1);

  GS2d0 = dim3(ceil(n / (float)BS2d.x), ceil(ntheta / (float)BS2d.y));
  GS3d0 = dim3(ceil(n / (float)BS3d.x), ceil(n / (float)BS3d.y),
               ceil(pnz / (float)BS3d.z));
  GS3d1 = dim3(ceil(2 * n / (float)BS3d.x), ceil(2 * n / (float)BS3d.y),
               ceil(pnz / (float)BS3d.z));
  GS3d2 = dim3(ceil((2 * n + 2 * m) / (float)BS3d.x),
               ceil((2 * n + 2 * m) / (float)BS3d.y), ceil(pnz / (float)BS3d.z));
  GS3d3 = dim3(ceil(n / (float)BS3d.x), ceil(ntheta / (float)BS3d.y),
               ceil(pnz / (float)BS3d.z));
}

// destructor, memory deallocation
radonusfft::~radonusfft() { free(); }

void radonusfft::free() {
  if (!is_free) {
    hipFree(f);
    hipFree(g);
    hipFree(fde);
    hipFree(fdee);
    hipFree(x);
    hipFree(y);
    hipFree(shiftfwd);
    hipFree(shiftadj);
    hipfftDestroy(plan2dfwd);
    hipfftDestroy(plan2dadj);
    hipfftDestroy(plan1d);
    is_free = true;
  }
}

void radonusfft::fwd(size_t g_, size_t f_) {
  hipMemcpy(f, (float2 *)f_, n * n * pnz * sizeof(float2), hipMemcpyDefault);
  hipMemset(fde, 0, 2 * n * 2 * n * pnz * sizeof(float2));
  hipMemset(fdee, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

  circ <<<GS3d0, BS3d>>> (f, 1.0f / n, n, pnz);
  takexy <<<GS2d0, BS2d>>> (x, y, theta, n, ntheta);

  divphi <<<GS3d0, BS3d>>> (fde, f, mu, n, pnz, TOMO_FWD);
  fftshiftc <<<GS3d1, BS3d>>> (fde, 2 * n, pnz);
  hipfftExecC2C(plan2dfwd, (hipfftComplex *)fde,
               (hipfftComplex *)&fdee[m + m * (2 * n + 2 * m)], HIPFFT_FORWARD);
  fftshiftc <<<GS3d2, BS3d>>> (fdee, 2 * n + 2 * m, pnz);

  wrap <<<GS3d2, BS3d>>> (fdee, n, pnz, m, TOMO_FWD);
  gather <<<GS3d3, BS3d>>> (g, fdee, x, y, m, mu, n, ntheta, pnz, TOMO_FWD);
  // shift with respect to given center
  shift <<<GS3d3, BS3d>>> (g, shiftfwd, n, ntheta, pnz);

  ifftshiftc <<<GS3d3, BS3d>>> (g, n, ntheta, pnz);
  hipfftExecC2C(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
  ifftshiftc <<<GS3d3, BS3d>>> (g, n, ntheta, pnz);

  hipMemcpy((float2 *)g_, g, n * ntheta * pnz * sizeof(float2),
             hipMemcpyDefault);
}

void radonusfft::adj(size_t f_, size_t g_) {
  hipMemcpy(g, (float2 *)g_, n * ntheta * pnz * sizeof(float2),
             hipMemcpyDefault);
  hipMemset(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));
  hipMemset(fdee, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

  takexy <<<GS2d0, BS2d>>> (x, y, theta, n, ntheta);

  ifftshiftc <<<GS3d3, BS3d>>> (g, n, ntheta, pnz);
  hipfftExecC2C(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
  ifftshiftc <<<GS3d3, BS3d>>> (g, n, ntheta, pnz);
  // applyfilter<<<GS3d3, BS3d>>>(g,n,ntheta,pnz);
  // shift with respect to given center
  shift <<<GS3d3, BS3d>>> (g, shiftadj, n, ntheta, pnz);

  gather <<<GS3d3, BS3d>>> (g, fdee, x, y, m, mu, n, ntheta, pnz, TOMO_ADJ);
  wrap <<<GS3d2, BS3d>>> (fdee, n, pnz, m, TOMO_ADJ);

  fftshiftc <<<GS3d2, BS3d>>> (fdee, 2 * n + 2 * m, pnz);
  hipfftExecC2C(plan2dadj, (hipfftComplex *)&fdee[m + m * (2 * n + 2 * m)],
               (hipfftComplex *)fde, HIPFFT_BACKWARD);
  fftshiftc <<<GS3d1, BS3d>>> (fde, 2 * n, pnz);

  divphi <<<GS3d0, BS3d>>> (fde, f, mu, n, pnz, TOMO_ADJ);
  circ <<<GS3d0, BS3d>>> (f, 1.0f / n, n, pnz);

  hipMemcpy((float2 *)f_, f, n * n * pnz * sizeof(float2),
             hipMemcpyDefault);
}
